/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void Convolution( long int *mat, long int *filter, long int* ans, int m, int n, int k){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ long int shared[];

    int local_row = threadIdx.y + k / 2;
    int local_col = threadIdx.x + k / 2;

    long int *shared_filter = shared;
    long int *shared_mat = shared + k * k;

    if (threadIdx.y < k && threadIdx.x < k) {
        shared_filter[threadIdx.y * k + threadIdx.x] = filter[threadIdx.y * k + threadIdx.x];
    }

    for (int i = -k / 2; i <= k / 2; ++i) {
        for (int j = -k / 2; j <= k / 2; ++j) {
            int r = row + i;
            int c = col + j;
            if (r >= 0 && r < m && c >= 0 && c < n) {
                shared_mat[(i + k / 2) * (k + 31) + (j + k / 2)] = mat[r * n + c];
            } else {
                shared_mat[(i + k / 2) * (k + 31) + (j + k / 2)] = 0;
            }
        }
    }

    __syncthreads();

    if (row < m && col < n) {
        long int sum = 0;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < k; ++j) {
                sum += shared_mat[(local_row + i) * (k + 31) + (local_col + j)] * shared_filter[i * k + j];
            }
        }
        ans[row * n + col] = sum;
    }
}


int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/

    long int* d_h_mat;
    long int* d_h_filter;
    long int* d_h_ans;

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid(ceil(n / 32.0), ceil(m / 32.0));

    int shared_mem_size = (k * k + (k + 31)*(k + 31)*sizeof(long int));

    hipMalloc(&d_h_mat, m*n*sizeof(long int));
    hipMemcpy(d_h_mat, h_mat, m*n*sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&d_h_filter, k*k*sizeof(long int));
    hipMemcpy(d_h_filter, h_filter, k*k*sizeof(long int), hipMemcpyHostToDevice);
    hipMalloc(&d_h_ans, m*n*sizeof(long int));
    
    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch

    Convolution<<< blocksPerGrid, threadsPerBlock >>>(d_h_mat, d_h_filter, d_h_ans, m, n, k);
    hipDeviceSynchronize(); 

    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch

    hipMemcpy(h_ans, d_h_ans, m*n*sizeof(long int), hipMemcpyDeviceToHost);

    hipFree(d_h_mat);
    hipFree(d_h_filter);
    hipFree(d_h_ans);
    
    
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}